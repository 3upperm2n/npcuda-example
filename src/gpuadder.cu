#include <device_kernel.cu>
#include <gpuadder.hh>
#include <assert.h>
#include <iostream>
using namespace std;

GPUAdder::GPUAdder (int* array_host_, int length_) {
  array_host = array_host_;
  length = length_;
  int size = length * sizeof(int);

  hipError_t err = hipMalloc((void**) &array_device, size);
  assert(err == 0);

  hipMemcpy(array_device, array_host, size, hipMemcpyHostToDevice);
}

void GPUAdder::increment() {
  kernel_add_one<<<1, length>>>(array_device);
}

void GPUAdder::retreive() {
  int size = length * sizeof(int);
  hipMemcpy(array_host, array_device, size, hipMemcpyDeviceToHost);
}

void GPUAdder::retreive_to (int* array_host_, int length_) {
    assert(length == length_);
    int size = length * sizeof(int);
    hipMemcpy(array_host_, array_device, size, hipMemcpyDeviceToHost);
}

GPUAdder::~GPUAdder() {
  hipFree(array_device);
}