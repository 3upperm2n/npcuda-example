
#include <hip/hip_runtime.h>
#include <stdio.h>

void __global__ kernel_add_one(int* a) {
    a[threadIdx.x] += 1;
}

int amain() {
    const int blocksize = 16;
    int a[blocksize] = {0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0};
    
    int *ad;
    int asize = blocksize*sizeof(int);

    hipMalloc( (void**) &ad, asize);
    hipMemcpy(ad, a, asize, hipMemcpyHostToDevice);
    
    dim3 dimBlock( blocksize, 1 );
    dim3 dimGrid( 1, 1 );    
    
    kernel_add_one<<<dimGrid, dimBlock>>>(ad);
    
    hipMemcpy(a, ad, asize, hipMemcpyDeviceToHost);
    hipFree(ad);
    
    printf("%d\n", a[0]);
    
    return 1;
}

