#include <kernel.cu>
#include <GPULearn.hh>
#include <assert.h>
#include <iostream>

#include <hip/hip_runtime_api.h>

using namespace std;

GPULearn::GPULearn(float* a_in, int len_a,
		           float* b_in, int len_b) 
{
	// check dim
	assert(len_a == len_b);

	a_h = a_in;
	b_h = b_in;

	length = len_a;

	size_t bytes = length * sizeof(float);


	// allocate device memory
	checkCudaErrors(hipMalloc(&a_d, bytes));
	checkCudaErrors(hipMalloc(&b_d, bytes));
	checkCudaErrors(hipMalloc(&c_d, bytes));

	//hipError_t err = hipMalloc((void**) &array_device, size);
	//assert(err == 0);

	checkCudaErrors(hipMemcpy(a_d, a_h, bytes, hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(b_d, b_h, bytes, hipMemcpyHostToDevice));

	//err = hipMemcpy(array_device, array_host, size, hipMemcpyHostToDevice);
	//assert(err == 0);
}

void GPULearn::vectorAdd() {

	dim3 blocks = dim3(256, 1, 1);
	dim3 grids  = dim3(BLK(length, 256), 1, 1);

	kernel_vectorAdd<<< grids, blocks >>>(a_d, b_d, c_d, length);
}

void GPULearn::getData() {
	size_t bytes = length * sizeof(float);
	checkCudaErrors(hipMemcpy(c_h, c_d, bytes, hipMemcpyDeviceToHost));
}

void GPULearn::getData_extern(float* c_out, int dim_c) {
	assert(length == dim_c);
	size_t bytes = length * sizeof(float);
	checkCudaErrors(hipMemcpy(c_out, c_d, bytes, hipMemcpyDeviceToHost));
}

GPULearn::~GPULearn() {
	checkCudaErrors(hipFree(a_d));
	checkCudaErrors(hipFree(b_d));
	checkCudaErrors(hipFree(c_d));
}
